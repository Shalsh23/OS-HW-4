#include "hip/hip_runtime.h"
#include <stdio.h>



typedef float (*op_func_t) (float, float);



__device__ float add_func (float x, float y)

{

    return x + y;

}



__device__ float mul_func (float x, float y)

{

    return x * y;

}



// Static pointers to device functions

__device__ op_func_t p_add_func = add_func;

__device__ op_func_t p_mul_func = mul_func;



__global__ void kernel( op_func_t op )

{

        printf("Result: %f\n", ( *op )( 1.0, 2.0 ) );

}



int main()

{



    op_func_t h_add_func;

    op_func_t h_mul_func;



    // Copy device function pointer to host side

    hipMemcpyFromSymbol( &h_mul_func, HIP_SYMBOL(p_mul_func), sizeof( op_func_t ) );

    hipMemcpyFromSymbol( &h_add_func, HIP_SYMBOL(p_add_func), sizeof( op_func_t ) );



    op_func_t d_myfunc = h_mul_func;



    kernel<<<1,1>>>( d_myfunc );



    hipDeviceSynchronize();



    return EXIT_SUCCESS;

}