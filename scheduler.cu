#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

typedef struct bag_elem
{
	void* (*func)(void*);
	void* arg;
} Bag_elem;

Bag_elem Bag[6][1000];

int sm_index[] = {0,0,0,0,0,0}; // index for 6 SMs

int taskAdd(void* (*func)(void*), void* arg, int sm)
{
	Bag[sm][sm_index[sm]].func = func;
	Bag[sm][sm_index[sm]].arg = arg;
}

extern "C" void schedule()
{

}

