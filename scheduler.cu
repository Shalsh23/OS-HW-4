#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define debug 1


unsigned int jobChunkArray[1000];
int jobChunkCounter = 0;
unsigned int SMC_workerCount[] = {0, 0, 0, 0, 0, 0}; // Array of counter for blocks created for each SM.

extern "C" __device__ void * square(void *func_arg)
{
	// __SMC_Begin
	//printf("The value of x is %d \n",x);
	//printf("The value of threadIdx.x is %d\n",threadIdx.x);
      //  printf("The value of blockIdx.x is %d\n", blockIdx.x);
	//printf("The value of blockIdx.x is %d\n",blockDim.x);
	

	//if (x > length)
	//	return;
	printf("In square\n.");
	int* array = (int*) func_arg;
	// int threadIdx = func_arg->threadIdx;
    int f = array[threadIdx.x];
    array[threadIdx.x] = f * f;
    return (void *)array;
    // __SMC_End
}

typedef void* (*func)(void *);

// Static pointers to device functions
__device__ func function;

typedef struct bag_elem
{
	//void* (*func)(void*);
	func y;
	void* arg;
} Bag_elem;

Bag_elem Bag[6][1000];

int sm_index[] = {0,0,0,0,0,0}; // index for 6 SMs

int jobchunk_id = 0;

int length_of_chunk = 6;

extern "C" int taskAdd(func userfunc, void* arg, int sm)
{
	if (debug) printf("In taskAdd:		Adding the task for sm = %d.\n", sm);
	function = userfunc;
	if (debug) printf("In taskAdd:		Adding the argument array for sm = %d. Bag's row_number is %d, column number is %d\n", sm, sm_index[sm], sm);
	(Bag[sm_index[sm]][sm]).arg = arg;
	int retval = jobchunk_id;
	jobchunk_id++;
	if (debug) printf("In taskAdd:		Adding to the jobChunkArray: jobChunkCounter: %d, retval: %d\n", jobChunkCounter, retval);
	jobChunkArray[jobChunkCounter] = retval; // Doubtful.
	jobChunkCounter++;
	return retval;
}


__global__ void persistent_func(Bag_elem Bag[6][1000], func d_user_func, unsigned int * __SMC_chunkCount, unsigned int * __SMC_newChunkSeq, unsigned int __SMC_chunksPerSM)
{
	//printf("In kernel:		Enter. Will start __SMC_Begin.\n");

	__shared__ int __SMC_workingCTAs;
	int __SMC_chunkID;
	uint __SMC_smid;
 	asm("mov.u32 %0, %smid;" : "=r"(__SMC_smid) );

 	int i;
		for (i = 0; i < 6; i++)
			printf("%u ",__SMC_chunkCount[i]);
		printf("\n");

	 printf("In kernel:		Got the Smid. It is %u.\n",__SMC_smid);


 	int offsetInCTA = threadIdx.x;
	if (offsetInCTA == 0)
    	__SMC_workingCTAs = atomicInc((unsigned int *)&__SMC_chunkCount[__SMC_smid], INT_MAX);

    printf("In kernel:		workingCTAs. It is %d.\n",__SMC_workingCTAs);

	__syncthreads(); 

	printf("In kernel:		workingCTAs. It is %d.\n",__SMC_workingCTAs);

	if (__SMC_workingCTAs >= __SMC_chunksPerSM)
    return;
	int __SMC_chunksPerCTA = __SMC_chunksPerSM / __SMC_chunksPerSM; 
	int __SMC_startChunkIDidx = __SMC_smid * __SMC_chunksPerSM + __SMC_workingCTAs * __SMC_chunksPerCTA;
	for (int __SMC_chunkIDidx = __SMC_startChunkIDidx; __SMC_chunkIDidx < __SMC_startChunkIDidx + __SMC_chunksPerCTA; __SMC_chunkIDidx ++) { 
    __SMC_chunkID = __SMC_newChunkSeq[__SMC_chunkIDidx];

	int x = threadIdx.x + __SMC_chunkID * blockDim.x;

	( *d_user_func )( (Bag[__SMC_smid][x]).arg );
	// (Bag[__SMC_smid][x])->y((Bag[__SMC_smid][x])->arg);
	
	}


}

extern "C" void schedule(int n, int m)
{
	if (debug) printf("In schedule:		Enter. Will copy device function pointer to host side.\n");

	func h_user_func;

	// Copy device function pointer to host side
	hipMemcpyFromSymbol( &h_user_func, HIP_SYMBOL(function), sizeof( func ) );

	if (debug) printf("In schedule:		Copying of the device function pointer done.\n");

	func d_user_func = h_user_func;

	// allocating memory on cuda for cpu variables
	Bag_elem d_Bag[6][1000];
	Bag_elem h_Bag[6][1000];

	if (debug) printf("In schedule:		TBD - Cuda Malloc for Bag in GPU.\n");

	// hipMalloc((void**) &__SMC_newChunkSeq, sizeof(int) * 1000);
	// hipMalloc((void**) &__SMC_workerCount, sizeof(Bag_elem)*6*1000);
	// hipMalloc((void**) &d_Bag, sizeof(Bag_elem)*6*1000);
	// hipMalloc((void**) &d_Bag, sizeof(Bag_elem)*6*1000);


	if (debug) printf("In schedule:		TBD - Cuda Memcpy for Bag in GPU.\n");
	// copying data from cpu to gpu
	hipMemcpy(d_Bag, Bag, sizeof(Bag_elem)*6*1000, hipMemcpyHostToDevice);

	if (debug) printf("In schedule:		SMC BEGIN.\n");

	if (debug)
	{
		int i;
		for (i = 0; i < 6; i++)
			printf("%d ",SMC_workerCount[i]);
		printf("\n");
	}


	unsigned int __SMC_workersNeeded = 2;  // Need to be made dynamic based on user input.
	unsigned int * __SMC_newChunkSeq = jobChunkArray;
	unsigned int * __SMC_workerCount = SMC_workerCount; // Array of counter for blocks created for each SM.


	if (debug)
	{
		int i;
		for (i = 0; i < 6; i++)
			printf("%u ",__SMC_workerCount[i]);
		printf("\n");
	}

	if (debug) printf("In schedule:		Kernel Call.\n");

	persistent_func <<< n, m >>> (d_Bag, d_user_func, __SMC_workerCount, __SMC_newChunkSeq, __SMC_workersNeeded);

	hipDeviceSynchronize();
	hipDeviceSynchronize();

	if (debug) printf("In schedule:		Kernel Call Ends. Do CudaMemCpy.\n");

	hipMemcpy(h_Bag, d_Bag, sizeof(Bag_elem)*6*1000, hipMemcpyDeviceToHost);

	if (debug) printf("In schedule:		Results copied into the CPU. Time for printing.\n");

	int i, y;
	int *temp_ans;
	for(i = 0; i < 6; i++)
	{
		temp_ans = (int *)h_Bag[i][0].arg;
		for (y = 0; y < length_of_chunk; y++)
			printf("%d ", temp_ans[y]);
		printf("\n");
	}
	hipFree(d_Bag);

}

