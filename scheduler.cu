#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

unsigned int* __SMC_buildChunkSeq()
{

}

#define __SMC_init  \
unsigned int * __SMC_workersNeeded = 2; \ //__SMC_numNeeded();  \
unsigned int * __SMC_newChunkSeq = __SMC_buildChunkSeq();  \
unsigned int * __SMC_workerCount = __SMC_initiateArray();

#define __SMC_Begin  \
__shared int __SMC_workingCTAs;  \
__SMC_getSMid;  \
if (offsetInCTA == 0)  \
    __SMC_workingCTAs = atomicInc(&__SMC_workerCount[__SMC_smid], INT_MAX);  \
__syncthreads();  \
if (__SMCS_workingCTAs >= __SMC_workersNeeded)  \
    return;  \
int __SMC_chunksPerCTA = \
    __SMC_chunksPerSM / __SMC_workersNeeded; \
int __SMC_startChunkIDidx = __SMC_smid * __SMC_chunksPerSM + \
    __SMC_workingCTAs * __SMC_chunksPerCTA;  \
for (int __SMC_chunkIDidx = __SMC_startChunkIDidx; \
    __SMC_chunkIDidx < __SMC_startChunkIDidx + \
        __SMC_chunksPerCTA; \
    __SMC_chunkDidx ++) {  \
    __SMC_chunkID = __SMC_newChunkSeq[__SMC_chunkIDidx]};

#define __SMC_End  }

#define __SMC_getSMid  \
uint __SMC_smid;  \
 asm("mov.u32 %0, %smid;" : "=r"(__SMC_smid) );



typedef struct bag_elem
{
	void* (*func)(void*);
	void* arg;
} Bag_elem;

Bag_elem Bag[6][1000];

int sm_index[] = {0,0,0,0,0,0}; // index for 6 SMs

int jobchunk_id = 0;

int taskAdd(void* (*func)(void*), void* arg, int sm)
{
	Bag[sm_index[sm]][sm].func = func;
	Bag[sm_index[sm]][sm].arg = arg;
	int retval = jobchunk_id;
	jobchunk_id++;
	return retval;
}

__global__ persistent_func(Bag_elem* Bag)
{

}

extern "C" void schedule()
{

	// allocating memory on cuda for cpu variables
	Bag_elem* d_Bag;
	hipMalloc((void**) &d_Bag, sizeof(Bag_elem)*6*1000);

	// copying data from cpu to gpu
	hipMemcpy(d_Bag, Bag, sizeof(Bag_elem)*6*1000, hipMemcpyHostToDevice);

	__SMC_init;

}

