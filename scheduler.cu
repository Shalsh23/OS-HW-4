#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


unsigned int jobChunkArray[1000];
int jobChunkCounter = 0;
unsigned int SMC_workerCount[] = {0, 0, 0, 0, 0, 0}; // Array of counter for blocks created for each SM.

// __device__ __host__ void square(long int * d_array_in, long int * d_array_out, int length)//, unsigned int * __SMC_chunkCount, unsigned int * __SMC_newChunkSeq, unsigned int __SMC_chunksPerSM)
// {
// 	// __SMC_Begin
// 	int x = threadIdx.x + blockIdx.x * blockDim.x;
// 	//printf("The value of x is %d \n",x);
// 	//printf("The value of threadIdx.x is %d\n",threadIdx.x);
//       //  printf("The value of blockIdx.x is %d\n", blockIdx.x);
// 	//printf("The value of blockIdx.x is %d\n",blockDim.x);
// 	if (x > length)
// 		return;
//     int f = d_array_in[x];
//     d_array_out[x] = f * f;
//     // __SMC_End
// }

// #define __SMC_init  \
// int __SMC_workersNeeded = 2; \ //__SMC_numNeeded();  // Need to be made dynamic based on user input.
// int * __SMC_newChunkSeq = jobChunkArray;  \
// int * __SMC_workerCount = SMC_workerCount; // Array of counter for blocks created for each SM.

// #define __SMC_Begin  \
// __shared int __SMC_workingCTAs;  \
// __SMC_getSMid;  \
// if (offsetInCTA == 0)  \
//     __SMC_workingCTAs = atomicInc(&__SMC_workerCount[__SMC_smid], INT_MAX);  \
// __syncthreads();  \
// if (__SMCS_workingCTAs >= __SMC_workersNeeded)  \
//     return;  \
// int __SMC_chunksPerCTA = \
//     __SMC_chunksPerSM / __SMC_workersNeeded; \
// int __SMC_startChunkIDidx = __SMC_smid * __SMC_chunksPerSM + \
//     __SMC_workingCTAs * __SMC_chunksPerCTA;  \
// for (int __SMC_chunkIDidx = __SMC_startChunkIDidx; \
//     __SMC_chunkIDidx < __SMC_startChunkIDidx + \
//         __SMC_chunksPerCTA; \
//     __SMC_chunkDidx ++) {  \
//     __SMC_chunkID = __SMC_newChunkSeq[__SMC_chunkIDidx]};

// #define __SMC_End  }

// #define __SMC_getSMid  \
// uint __SMC_smid;  \
//  asm("mov.u32 %0, %smid;" : "=r"(__SMC_smid) );

// typedef struct func_arg
// {
// 	int* arg_arr;
// 	int threadIdx;
// } f_arg;

// __device__ f_arg farg_dev; // device copy of struct f_arg





extern "C" __device__ void * square(void *func_arg)
{
	// __SMC_Begin
	//printf("The value of x is %d \n",x);
	//printf("The value of threadIdx.x is %d\n",threadIdx.x);
      //  printf("The value of blockIdx.x is %d\n", blockIdx.x);
	//printf("The value of blockIdx.x is %d\n",blockDim.x);
	

	//if (x > length)
	//	return;
	int* array = (int*) func_arg;
	// int threadIdx = func_arg->threadIdx;
    int f = array[threadIdx.x];
    array[threadIdx.x] = f * f;
    return (void *)array;
    // __SMC_End
}

typedef void* (*func)(void *);

// Static pointers to device functions
__device__ func function;

typedef struct bag_elem
{
	//void* (*func)(void*);
	func y;
	void* arg;
} Bag_elem;

Bag_elem Bag[6][1000];

int sm_index[] = {0,0,0,0,0,0}; // index for 6 SMs

int jobchunk_id = 0;

int length_of_chunk = 6;

extern "C" int taskAdd(func userfunc, void* arg, int sm)
{
	// (Bag[sm_index[sm]][sm]).y = square;
	function = userfunc;
	(Bag[sm_index[sm]][sm]).arg = arg;
	int retval = jobchunk_id;
	jobchunk_id++;
	jobChunkArray[jobChunkCounter] = retval; // Doubtful.
	jobChunkCounter++;
	return retval;
}


__global__ void persistent_func(Bag_elem (*Bag)[1000], func d_user_func, unsigned int * __SMC_chunkCount, unsigned int * __SMC_newChunkSeq, unsigned int __SMC_chunksPerSM)
{

	__shared__ int __SMC_workingCTAs;
	int __SMC_chunkID;
	uint __SMC_smid;  \
 	asm("mov.u32 %0, %smid;" : "=r"(__SMC_smid) );
	//if (offsetInCTA == 0)
    __SMC_workingCTAs = atomicInc(&__SMC_chunkCount[__SMC_smid], INT_MAX);
	__syncthreads(); 
	if (__SMC_workingCTAs >= __SMC_chunksPerSM)
    return;
	int __SMC_chunksPerCTA = __SMC_chunksPerSM / __SMC_chunksPerSM; 
	int __SMC_startChunkIDidx = __SMC_smid * __SMC_chunksPerSM + __SMC_workingCTAs * __SMC_chunksPerCTA;
	for (int __SMC_chunkIDidx = __SMC_startChunkIDidx; __SMC_chunkIDidx < __SMC_startChunkIDidx + __SMC_chunksPerCTA; __SMC_chunkIDidx ++) { 
    __SMC_chunkID = __SMC_newChunkSeq[__SMC_chunkIDidx];

	int x = threadIdx.x + __SMC_chunkID * blockDim.x;

	( *d_user_func )( (Bag[__SMC_smid][x]).arg );
	// (Bag[__SMC_smid][x])->y((Bag[__SMC_smid][x])->arg);
	
	}


}

extern "C" void schedule(int n, int m)
{
	//farg_dev = ;

	func h_user_func;

	// Copy device function pointer to host side
	hipMemcpyFromSymbol( &h_user_func, HIP_SYMBOL(function), sizeof( func ) );

	//f_arg h_f_arg;
	//cudaMemcpyFromSymbol( &h_f_arg, farg_dev, sizeof( f_arg ) );

	//f_arg d_f_arg = h_f_arg;

	func d_user_func = h_user_func;

	// allocating memory on cuda for cpu variables
	Bag_elem d_Bag[6][1000];
	Bag_elem h_Bag[6][1000];

	hipMalloc((void**) &d_Bag, sizeof(Bag_elem)*6*1000);

	// copying data from cpu to gpu
	hipMemcpy(&d_Bag, &Bag, sizeof(Bag_elem)*6*1000, hipMemcpyHostToDevice);

	unsigned int __SMC_workersNeeded = 2;  //__SMC_numNeeded();  // Need to be made dynamic based on user input.
	unsigned int * __SMC_newChunkSeq = jobChunkArray;
	unsigned int * __SMC_workerCount = SMC_workerCount; // Array of counter for blocks created for each SM.

	persistent_func <<< n, m >>> (d_Bag, d_user_func, __SMC_workerCount, __SMC_newChunkSeq, __SMC_workersNeeded);

	hipMemcpy(&h_Bag, &d_Bag, sizeof(Bag_elem)*6*1000, hipMemcpyDeviceToHost);

	int i, y;
	int *temp_ans;
	for(i = 0; i < 6; i++)
	{
		temp_ans = (int *)h_Bag[i][0].arg;
		for (y = 0; y < length_of_chunk; y++)
			printf("%d ", temp_ans[y]);
		printf("\n");
	}
	hipFree(d_Bag);

}

