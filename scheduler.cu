#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


unsigned int jobChunkArray[1000];
int jobChunkCounter = 0;
unsigned int SMC_workerCount[] = {0, 0, 0, 0, 0, 0}; // Array of counter for blocks created for each SM.

// __device__ __host__ void square(long int * d_array_in, long int * d_array_out, int length)//, unsigned int * __SMC_chunkCount, unsigned int * __SMC_newChunkSeq, unsigned int __SMC_chunksPerSM)
// {
// 	// __SMC_Begin
// 	int x = threadIdx.x + blockIdx.x * blockDim.x;
// 	//printf("The value of x is %d \n",x);
// 	//printf("The value of threadIdx.x is %d\n",threadIdx.x);
//       //  printf("The value of blockIdx.x is %d\n", blockIdx.x);
// 	//printf("The value of blockIdx.x is %d\n",blockDim.x);
// 	if (x > length)
// 		return;
//     int f = d_array_in[x];
//     d_array_out[x] = f * f;
//     // __SMC_End
// }

// #define __SMC_init  \
// int __SMC_workersNeeded = 2; \ //__SMC_numNeeded();  // Need to be made dynamic based on user input.
// int * __SMC_newChunkSeq = jobChunkArray;  \
// int * __SMC_workerCount = SMC_workerCount; // Array of counter for blocks created for each SM.

// #define __SMC_Begin  \
// __shared int __SMC_workingCTAs;  \
// __SMC_getSMid;  \
// if (offsetInCTA == 0)  \
//     __SMC_workingCTAs = atomicInc(&__SMC_workerCount[__SMC_smid], INT_MAX);  \
// __syncthreads();  \
// if (__SMCS_workingCTAs >= __SMC_workersNeeded)  \
//     return;  \
// int __SMC_chunksPerCTA = \
//     __SMC_chunksPerSM / __SMC_workersNeeded; \
// int __SMC_startChunkIDidx = __SMC_smid * __SMC_chunksPerSM + \
//     __SMC_workingCTAs * __SMC_chunksPerCTA;  \
// for (int __SMC_chunkIDidx = __SMC_startChunkIDidx; \
//     __SMC_chunkIDidx < __SMC_startChunkIDidx + \
//         __SMC_chunksPerCTA; \
//     __SMC_chunkDidx ++) {  \
//     __SMC_chunkID = __SMC_newChunkSeq[__SMC_chunkIDidx]};

// #define __SMC_End  }

// #define __SMC_getSMid  \
// uint __SMC_smid;  \
//  asm("mov.u32 %0, %smid;" : "=r"(__SMC_smid) );

__device__ void * square(void *arr)
{
	// __SMC_Begin
	//printf("The value of x is %d \n",x);
	//printf("The value of threadIdx.x is %d\n",threadIdx.x);
      //  printf("The value of blockIdx.x is %d\n", blockIdx.x);
	//printf("The value of blockIdx.x is %d\n",blockDim.x);
	

	//if (x > length)
	//	return;
	int* array = (int*) arr;
    int f = array[threadIdx.x];
    array[threadIdx.x] = f * f;
    return (void *)array;
    // __SMC_End
}

typedef void* (*func)(void *);

typedef struct bag_elem
{
	//void* (*func)(void*);
	func y;
	void* arg;
} Bag_elem;

Bag_elem Bag[6][1000];

int sm_index[] = {0,0,0,0,0,0}; // index for 6 SMs

int jobchunk_id = 0;

int length_of_chunk = 6;

int taskAdd(void* (*func)(void*), void* arg, int sm)
{
	(Bag[sm_index[sm]][sm]).y = square;
	(Bag[sm_index[sm]][sm]).arg = arg;
	int retval = jobchunk_id;
	jobchunk_id++;
	jobChunkArray[jobChunkCounter] = retval; // Doubtful.
	jobChunkCounter++;
	return retval;
}


__global__ void persistent_func(Bag_elem* Bag, unsigned int * __SMC_chunkCount, unsigned int * __SMC_newChunkSeq, unsigned int __SMC_chunksPerSM)
{
	__shared__ int __SMC_workingCTAs;
	int __SMC_chunkID;
	uint __SMC_smid;  \
 	asm("mov.u32 %0, %smid;" : "=r"(__SMC_smid) );
	//if (offsetInCTA == 0)
    __SMC_workingCTAs = atomicInc(&__SMC_chunkCount[__SMC_smid], INT_MAX);
	__syncthreads(); 
	if (__SMC_workingCTAs >= __SMC_chunksPerSM)
    return;
	int __SMC_chunksPerCTA = __SMC_chunksPerSM / __SMC_chunksPerSM; 
	int __SMC_startChunkIDidx = __SMC_smid * __SMC_chunksPerSM + __SMC_workingCTAs * __SMC_chunksPerCTA;
	for (int __SMC_chunkIDidx = __SMC_startChunkIDidx; __SMC_chunkIDidx < __SMC_startChunkIDidx + __SMC_chunksPerCTA; __SMC_chunkIDidx ++) { 
    __SMC_chunkID = __SMC_newChunkSeq[__SMC_chunkIDidx];

	int x = threadIdx.x + __SMC_chunkID * blockDim.x;

	// (Bag[__SMC_smid][x])->y((Bag[__SMC_smid][x])->arg);
	
	}


}

extern "C" void schedule(int n, int m)
{
	// allocating memory on cuda for cpu variables
	Bag_elem* d_Bag;
	hipMalloc((void**) &d_Bag, sizeof(Bag_elem)*6*1000);

	// copying data from cpu to gpu
	hipMemcpy(d_Bag, Bag, sizeof(Bag_elem)*6*1000, hipMemcpyHostToDevice);

	unsigned int __SMC_workersNeeded = 2;  //__SMC_numNeeded();  // Need to be made dynamic based on user input.
	unsigned int * __SMC_newChunkSeq = jobChunkArray;
	unsigned int * __SMC_workerCount = SMC_workerCount; // Array of counter for blocks created for each SM.

	persistent_func <<< n, m >>> (d_Bag, __SMC_workerCount, __SMC_newChunkSeq, __SMC_workersNeeded);

	hipMemcpyAsync(Bag, d_Bag, sizeof(Bag_elem)*6*1000, hipMemcpyDeviceToHost);

	int i, y;
	for(i = 0; i < 6; i++)
	{
		for (y = 0; y < length_of_chunk; y++)
			// printf("%d ", (Bag[0][i].arg)[y]);
		printf("\n");
	}
	hipFree(d_Bag);

}

