#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define debug 1

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

unsigned int jobChunkArray[] = {0, 1, 2, 3, 4, 5, 6 ,7 , 8, 9, 10, 11};
int jobChunkCounter = 0;
unsigned int SMC_workerCount[] = {0, 0, 0, 0, 0, 0}; // Array of counter for blocks created for each SM.

extern "C" __device__ void * square(void *number)
{

	int* array = (int*) number;
	//printf("Inside square\n");
	//printf("Before Square:		%d\n", *array);
    *array *= *array;
    //printf("After Square:		%d\n", *array);
    return (void *)array;
    // __SMC_End
}

typedef void* (*func)(void *);

// Static pointers to device functions
__device__ func function = square;

/*typedef struct bag_elem
{
	//void* (*func)(void*);
	func y;
	void* arg;
} Bag_elem;
*/
//Bag_elem Bag[6000];
int Bag[6000];

int sm_index[] = {0,0,0,0,0,0}; // index for 6 SMs

int jobchunk_id = 0;

int length_of_chunk = 6;

int j = 0;
extern "C" int taskAdd(func userfunc, void* arg, int sm)
{
	if (debug) printf("In taskAdd:		Adding the task for sm = %d.\n", sm);
	//function = userfunc;
	if (debug) printf("In taskAdd:		Adding the argument array for sm = %d. Bag's row_number is %d, column number is %d\n", sm, sm_index[sm], sm);
	int i = 0;
	// if (j == 6)
	// 	j = 0;
	int * argv = (int *)arg;
	for(i = 0; i < 6; i++)
		Bag[(sm * 1000) + i] = argv[i];
	int retval = jobchunk_id;
	jobchunk_id++;
	if (debug) printf("In taskAdd:		Adding to the jobChunkArray: jobChunkCounter: %d, retval: %d\n", jobChunkCounter, retval);
	// jobChunkArray[jobChunkCounter] = retval; // Doubtful.
	// jobChunkCounter++;
	// jobChunkArray[jobChunkCounter] = retval + 1; // Hardcode.

	return retval;
}


__global__ void persistent_func(int *d_Bag, func op, unsigned int * __SMC_chunkCount, unsigned int * __SMC_newChunkSeq, unsigned int __SMC_chunksPerSM)
{
	printf("In kernel:		Enter. Will start __SMC_Begin.\n");
	int *temp, i, y;
	__shared__ int __SMC_workingCTAs;
	int __SMC_chunkID;
	uint __SMC_smid;
 	asm("mov.u32 %0, %smid;" : "=r"(__SMC_smid) );
		// for (i = 0; i < 6; i++)
		// 	printf("%u ",__SMC_chunkCount[i]);
		// printf("\n");
 // 	for(i = 0; i < 6; i++)
	// {
	// 	printf("%d\n", d_Bag[__SMC_smid * 1000 + i]);
	// }
	printf("In kernel:		Got the Smid. It is %u.\n",__SMC_smid);


 	int offsetInCTA = threadIdx.x;
	if (offsetInCTA == 0)
    {
    	__SMC_workingCTAs = atomicInc((unsigned int *)&__SMC_chunkCount[__SMC_smid], INT_MAX);
		printf("In kernel:		workingCTAs. It is %d. Block is %d. SMID is %d\n",__SMC_workingCTAs, blockIdx.x, __SMC_smid);
	}

	__syncthreads(); 

	printf("In kernel:		__SMC_chunksPerSM is %d. __SMC_workingCTAs is %d. SMID is %d.\n",__SMC_chunksPerSM, __SMC_workingCTAs, __SMC_smid);

	if (__SMC_workingCTAs >= __SMC_chunksPerSM)
    return;

	int __SMC_chunksPerCTA = __SMC_chunksPerSM / __SMC_chunksPerSM; 
	int __SMC_startChunkIDidx = __SMC_smid * __SMC_chunksPerSM + __SMC_workingCTAs * __SMC_chunksPerCTA;

	printf("In kernel:		__SMC_startChunkIDidx is %d. SMID is %d.\n", __SMC_startChunkIDidx, __SMC_smid);
	for (int __SMC_chunkIDidx = __SMC_startChunkIDidx; __SMC_chunkIDidx < __SMC_startChunkIDidx + __SMC_chunksPerCTA; __SMC_chunkIDidx ++) { 
	    printf("SMID is %d. __SMC_chunkIDidx is %d. __SMC_newChunkSeq[__SMC_chunkIDidx] is %d.\n", __SMC_smid,__SMC_chunkIDidx, __SMC_newChunkSeq[__SMC_chunkIDidx]);

	    __SMC_chunkID = __SMC_newChunkSeq[__SMC_chunkIDidx];
	    //printf("chunk id \n%d\n", __SMC_chunkID);
		int x = threadIdx.x + __SMC_chunkID * blockDim.x;
	    printf("ThreadId is %d. __SMC_ChunkId is %d. blockDim.x is %d. x = %d\n", threadIdx.x, __SMC_chunkID, blockDim.x,x);

		//printf("Inside loop\n");
		// square((Bag[__SMC_smid][x]).arg );
		//int anum[] = {1,2};
		//int *bagval = (int*)d_Bag[__SMC_smid * 1000];
		int x_coordinate = x / 6;
		int y_coordinate = x % 6;

		printf("Before Square %d ", d_Bag[x_coordinate * 1000 + y_coordinate]);
		square(&d_Bag[x_coordinate * 1000 + y_coordinate]);
		printf("After Square %d\n", d_Bag[x_coordinate * 1000 + y_coordinate]);

		// (Bag[__SMC_smid][x])->y((Bag[__SMC_smid][x])->arg);
		printf("After function call\n");
	}


}

extern "C" void schedule(int n, int m)
{

	int *temp, i,y;
	if (debug) printf("In schedule:		Enter. Will copy device function pointer to host side.\n");

	func h_user_func;
	unsigned int * __SMC_newChunkSeq;
	unsigned int * __SMC_chunkCount;

	// // Copy device function pointer to host side
	// hipMemcpyFromSymbol( &h_user_func, HIP_SYMBOL(function), sizeof( func ) );

	if (debug) printf("In schedule:		Copying of the device function pointer done.\n");

	func d_user_func = h_user_func;

	// allocating memory on cuda for cpu variables
	//Bag_elem d_Bag[6][1000];
	//Bag_elem h_Bag[6][1000];
	int *d_Bag_1d;
	int h_Bag_1d[6000];

	int *d_array;

	int * Bag_ptr = Bag;
	for(i = 0; i < 6; i++)
	{	
		//temp = (int *)Bag[1000 * i];
		for (y = 0; y < 6; y++)
			printf("%d ", Bag[i * 1000 + y]);
		printf("\n");
	}

	if (debug) printf("In schedule:		TBD - Cuda Malloc for Bag in GPU.\n");

	// hipMalloc((void**) &__SMC_newChunkSeq, sizeof(int) * 1000);
	// hipMalloc((void**) &__SMC_workerCount, sizeof(Bag_elem)*6*1000);
	// hipMalloc((void**) &d_Bag, sizeof(Bag_elem)*6*1000);
	printf("Hello\n");
	hipError_t rc = hipMalloc((void**) &d_Bag_1d, sizeof(int)*6000);
	

	if (rc != hipSuccess)
    	printf("Could not allocate memory: %d", rc);
	printf("Hello1\n");

	hipMalloc((void**) &__SMC_chunkCount, sizeof(int)*1000);
		
	printf("Hello2\n");

	hipMalloc((void**) &__SMC_newChunkSeq, sizeof(int)*12);


	if (debug) printf("In schedule:		TBD - Cuda Memcpy for Bag in GPU.\n");
	// copying data from cpu to gpu
	// hipMemcpy(d_Bag, Bag, sizeof(Bag_elem)*6*1000, hipMemcpyHostToDevice);
	hipMemcpy(d_Bag_1d, Bag, sizeof(int)*6*1000, hipMemcpyHostToDevice);

	// for (i = 0; i < 6; i++)
	// {
	// 	gpuErrchk( hipMalloc((void**) &(d_array), sizeof(int)*6) );

	// 	hipMemcpy(d_array, Bag_ptr[i].arg, sizeof(int) * 6, hipMemcpyHostToDevice);

	// 	Bag_ptr[i].arg = d_array;
	// }
	// for(i = 0; i < 6; i++)
	// {	
	// 	temp = (int *)((Bag_ptr[1000 * i]).arg);
	// 	hipMemcpy(d_Bag_1d[1000 * i].arg, Bag_ptr[1000 * i].arg, sizeof(int) * 6, hipMemcpyHostToDevice);
	// }


	hipMemcpy(__SMC_chunkCount, SMC_workerCount, sizeof(int)*6, hipMemcpyHostToDevice);
	hipMemcpy(__SMC_newChunkSeq, jobChunkArray, sizeof(int)*12, hipMemcpyHostToDevice);

	//hipMemcpy(d_Bag, Bag, sizeof(Bag_elem)*6*1000, hipMemcpyHostToDevice);


	if (debug) printf("In schedule:		SMC BEGIN.\n");



		for (i = 0; i < 6000; i++)
			printf("%d ",Bag[i]);
		printf("\n");



	// unsigned int __SMC_workersNeeded = 2;  // Need to be made dynamic based on user input.
	// unsigned int * __SMC_newChunkSeq = jobChunkArray;
	// unsigned int * __SMC_workerCount = SMC_workerCount; // Array of counter for blocks created for each SM.

	if (debug) printf("In schedule:		Kernel Call.\n");

//unsigned int * __SMC_chunkCount, unsigned int * __SMC_newChunkSeq, unsigned int __SMC_chunksPerSM
	// persistent_func <<< n, m >>> (d_Bag, d_user_func, __SMC_chunkCount, __SMC_newChunkSeq, 2);
	persistent_func <<< n, m >>> (d_Bag_1d, d_user_func, __SMC_chunkCount, __SMC_newChunkSeq, 2);

	hipDeviceSynchronize();
	hipDeviceSynchronize();

	if (debug) printf("In schedule:		Kernel Call Ends. Do CudaMemCpy.\n");

	hipMemcpyAsync(h_Bag_1d, d_Bag_1d, sizeof(int)*6*1000, hipMemcpyDeviceToHost);

	if (debug) printf("In schedule:		Results copied into the CPU. Time for printing.\n");

	int *temp_ans;
	for(i = 0; i < 6000; i++)
	{
		printf("%d ", h_Bag_1d[i]);
	}
	printf("\n");

}

