#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define debug 1


unsigned int jobChunkArray[1000];
int jobChunkCounter = 0;
unsigned int SMC_workerCount[] = {0, 0, 0, 0, 0, 0}; // Array of counter for blocks created for each SM.

extern "C" __device__ void * square(void *func_arg)
{
	// __SMC_Begin
	//printf("The value of x is %d \n",x);
	//printf("The value of threadIdx.x is %d\n",threadIdx.x);
      //  printf("The value of blockIdx.x is %d\n", blockIdx.x);
	//printf("The value of blockIdx.x is %d\n",blockDim.x);
	

	//if (x > length)
	//	return;
	int* array = (int*) func_arg;
	// int threadIdx = func_arg->threadIdx;
    int f = array[threadIdx.x];
    array[threadIdx.x] = f * f;
    return (void *)array;
    // __SMC_End
}

typedef void* (*func)(void *);

// Static pointers to device functions
__device__ func function;

typedef struct bag_elem
{
	//void* (*func)(void*);
	func y;
	void* arg;
} Bag_elem;

Bag_elem Bag[6][1000];

int sm_index[] = {0,0,0,0,0,0}; // index for 6 SMs

int jobchunk_id = 0;

int length_of_chunk = 6;

extern "C" int taskAdd(func userfunc, void* arg, int sm)
{
	if (debug) printf("In taskAdd:		Adding the task for sm = %d.\n", sm);
	function = userfunc;
	if (debug) printf("In taskAdd:		Adding the argument array for sm = %d. Bag's row_number is %d, column number is %d\n", sm, sm_index[sm], sm);
	(Bag[sm_index[sm]][sm]).arg = arg;
	int retval = jobchunk_id;
	jobchunk_id++;
	if (debug) printf("In taskAdd:		Adding to the jobChunkArray: jobChunkCounter: %d, retval: %d\n", jobChunkCounter, retval);
	jobChunkArray[jobChunkCounter] = retval; // Doubtful.
	jobChunkCounter++;
	return retval;
}


__global__ void persistent_func(Bag_elem (*Bag)[1000], func d_user_func, unsigned int * __SMC_chunkCount, unsigned int * __SMC_newChunkSeq, unsigned int __SMC_chunksPerSM)
{

	__shared__ int __SMC_workingCTAs;
	int __SMC_chunkID;
	uint __SMC_smid;
 	asm("mov.u32 %0, %smid;" : "=r"(__SMC_smid) );
	//if (offsetInCTA == 0)
    __SMC_workingCTAs = atomicInc(&__SMC_chunkCount[__SMC_smid], INT_MAX);
	__syncthreads(); 
	if (__SMC_workingCTAs >= __SMC_chunksPerSM)
    return;
	int __SMC_chunksPerCTA = __SMC_chunksPerSM / __SMC_chunksPerSM; 
	int __SMC_startChunkIDidx = __SMC_smid * __SMC_chunksPerSM + __SMC_workingCTAs * __SMC_chunksPerCTA;
	for (int __SMC_chunkIDidx = __SMC_startChunkIDidx; __SMC_chunkIDidx < __SMC_startChunkIDidx + __SMC_chunksPerCTA; __SMC_chunkIDidx ++) { 
    __SMC_chunkID = __SMC_newChunkSeq[__SMC_chunkIDidx];

	int x = threadIdx.x + __SMC_chunkID * blockDim.x;

	( *d_user_func )( (Bag[__SMC_smid][x]).arg );
	// (Bag[__SMC_smid][x])->y((Bag[__SMC_smid][x])->arg);
	
	}


}

extern "C" void schedule(int n, int m)
{
	if (debug) printf("In schedule:		Enter. Will copy device function pointer to host side.\n", sm);

	func h_user_func;

	// Copy device function pointer to host side
	hipMemcpyFromSymbol( &h_user_func, HIP_SYMBOL(function), sizeof( func ) );

	//f_arg h_f_arg;
	//hipMemcpyFromSymbol( &h_f_arg, HIP_SYMBOL(farg_dev), sizeof( f_arg ) );

	//f_arg d_f_arg = h_f_arg;

	func d_user_func = h_user_func;

	// allocating memory on cuda for cpu variables
	Bag_elem d_Bag[6][1000];
	Bag_elem h_Bag[6][1000];

	hipMalloc((void**) &d_Bag, sizeof(Bag_elem)*6*1000);

	// copying data from cpu to gpu
	hipMemcpy(d_Bag, Bag, sizeof(Bag_elem)*6*1000, hipMemcpyHostToDevice);

	unsigned int __SMC_workersNeeded = 2;  // Need to be made dynamic based on user input.
	unsigned int * __SMC_newChunkSeq = jobChunkArray;
	unsigned int * __SMC_workerCount = SMC_workerCount; // Array of counter for blocks created for each SM.

	persistent_func <<< n, m >>> (d_Bag, d_user_func, __SMC_workerCount, __SMC_newChunkSeq, __SMC_workersNeeded);

	hipMemcpy(h_Bag, d_Bag, sizeof(Bag_elem)*6*1000, hipMemcpyDeviceToHost);

	int i, y;
	int *temp_ans;
	for(i = 0; i < 6; i++)
	{
		temp_ans = (int *)h_Bag[i][0].arg;
		for (y = 0; y < length_of_chunk; y++)
			printf("%d ", temp_ans[y]);
		printf("\n");
	}
	hipFree(d_Bag);

}

